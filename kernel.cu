#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h> 
#include <thrust/fill.h>
#include <thrust/replace.h> 
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
using namespace std;

#include "cuda/cuGlobal.h"
#include "cuda/deviceFuncs.h"
#include "cuda/cuWrap.h"
#include "cuda/cuRealise.h"



