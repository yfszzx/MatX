#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hiprand.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/copy.h> 
#include <thrust/fill.h>
#include <thrust/replace.h> 
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
using namespace std;

#include "cuGlobal.h"
#include "deviceFuncs.h"
#include "cuWrap.h"
#include "cuRealise.h"



